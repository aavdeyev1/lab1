#include <stdio.h>
#include <stdlib.h>
#include <math.h>
// #include <time.h>
#include <pthread.h>
#include "hip/hip_runtime.h"

 
typedef unsigned long long bignum;
__device__ int isPrimeGPU(bignum x);

// CUDA kernel. Each thread takes care of one element of c
__global__ void primeGPU(bignum *a, bignum *result, int n)
{
    // Get our global thread ID
    int id = blockIdx.x*blockDim.x+threadIdx.x;
 
    // Make sure we do not go out of bounds
    if (id < n)
        result[id] = isPrimeGPU(a[id]);
}

__device__ int isPrimeGPU(bignum x) {

   bignum i;
   for (i = 2; i * i < x + 1; i++) {
       if (x % i == 0) {
           return 0;
       }
   }
   return 1;
}



void printArray(bignum * a, int len){
 
    int i;
    printf("\n[");
    for(i=0; i<len; i++){
    
       printf("%llu, ", a[i]);
 
    }
    printf("]\n");
 
 }
 
int main( int argc, char* argv[] )
{
    
    if(argc < 3)
    {
        printf("Usage: too few arguments\n");
        exit(-1);
    }
    // Retrieve N, blockSize from args
    bignum N = (bignum) (atoi(argv[1]) + 1);

    int odds = (int)ceil((double)(N + 1)/2);
    size_t bytes = (bignum)(odds * sizeof(bignum));

    bignum *h_input;
    bignum *h_output;

    h_input = (bignum *)malloc(bytes);
    h_output = (bignum *)malloc(bytes);

    bignum *d_input;
    bignum *d_output;

    hipMalloc(&d_input, bytes);
    hipMalloc(&d_output, bytes);

   //  int odds = ceil((double)(N + 1)/2)

    int i, s;
    for (i=0; i < N + 1; i++){
      if (s % 2 != 0){
         h_input[i] = s;
         h_output[i] = 0;    
      }
      s++;
      // h_input[i] = i;
      // h_output[i] = 0;    
    }
    printArray(h_input, odds);
    printArray(h_output, odds);

    hipMemcpy( d_input, h_input, bytes, hipMemcpyHostToDevice);

    int blockSize, gridSize;
 
    // Number of threads in each thread block
    blockSize = (int) atoi(argv[2]);
 
    // Number of thread blocks in grid
    gridSize = (int)ceil((double)((double)((N)/2)/blockSize));
 
    // Execute the kernel
    primeGPU<<<gridSize, blockSize>>>(d_input, d_output, odds);
 
    // Copy array back to host
    hipMemcpy( h_output, d_output, bytes, hipMemcpyDeviceToHost );

    printArray(h_output, odds);
    
    free(h_input);
    free(h_output);
    hipFree(d_input);
    hipFree(d_output);


   //  cudaDeviceReset();

    return 0;
}
