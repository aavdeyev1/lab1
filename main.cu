#include <stdio.h>
#include <stdlib.h>
#include <math.h>
// #include <time.h>
#include <pthread.h>
#include "hip/hip_runtime.h"

 
typedef unsigned long long bignum;
__device__ int isPrimeGPU(bignum x);

// CUDA kernel. Each thread takes care of one element of c
__global__ void primeGPU(bignum *a, bignum *result, int n)
{
    // Get our global thread ID
    int id = blockIdx.x*blockDim.x+threadIdx.x;
 
    // Make sure we do not go out of bounds
    if (id < n && id > 2 ) {
      if (id % 2 != 0)
        result[id] = isPrimeGPU(a[id]);
    }
}

__device__ int isPrimeGPU(bignum x) {

   bignum i;
   for (i = 2; i * i < x + 1; i++) {
       if (x % i == 0) {
           return 0;
       }
   }
   return 1;
}



void printArray(bignum * a, int len){
 
    int i;
    printf("\n[");
    for(i=0; i<len; i++){
    
       printf("%llu, ", a[i]);
 
    }
    printf("]\n");
 
 }
 
int main( int argc, char* argv[] )
{
    
    if(argc < 3)
    {
        printf("Usage: too few arguments\n");
        exit(-1);
    }
    // Retrieve N, blockSize from args
    bignum N = (bignum) (atoi(argv[1]) + 1);

   //  int odds = (int)ceil((double)(N + 1)/2);
    size_t bytes = (bignum)(N * sizeof(bignum));

    bignum *h_input;
    bignum *h_output;

    h_input = (bignum *)malloc(bytes);
    h_output = (bignum *)malloc(bytes);

    bignum *d_input;
    bignum *d_output;

    hipMalloc(&d_input, bytes);
    hipMalloc(&d_output, bytes);

   //  int odds = ceil((double)(N + 1)/2)

    int i;
    for (i=0; i < N; i++){
      h_input[i] = i;
      h_output[i] = i; 
    }
    printArray(h_input, N);
    printArray(h_output, N);

    hipMemcpy( d_input, h_input, bytes, hipMemcpyHostToDevice);

    int blockSize, gridSize;
 
    // Number of threads in each thread block
    blockSize = (int) atoi(argv[2]);
 
    // Number of thread blocks in grid
    gridSize = (int)ceil((double)((double)((N)/2)/blockSize));
 
    // Execute the kernel
    primeGPU<<<gridSize, blockSize>>>(d_input, d_output, N);
 
    // Copy array back to host
    hipMemcpy( h_output, d_output, bytes, hipMemcpyDeviceToHost );

    printArray(h_output, N);

    int total = 0;
    for (i=0; i < N; i++){
      total = total + h_output[i];
    }
    printf("Number of primes in that range: %d\n", total);
    
    free(h_input);
    free(h_output);
    hipFree(d_input);
    hipFree(d_output);


   //  cudaDeviceReset();

    return 0;
}
